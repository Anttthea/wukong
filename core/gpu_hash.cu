#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016 Shanghai Jiao Tong University.
 *     All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing,
 *  software distributed under the License is distributed on an "AS
 *  IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either
 *  express or implied.  See the License for the specific language
 *  governing permissions and limitations under the License.
 *
 * For more about this software visit:
 *
 *      http://ipads.se.sjtu.edu.cn/projects/wukong
 *
 */

// #include "rdf_meta.hpp"
#include "gpu_hash.hpp"
#include <thrust/execution_policy.h>

/*********************************************
 *                                           *
 *                Utilities                  *
 *                                           *
 *********************************************/
enum { NBITS_DIR = 1 };
enum { NBITS_IDX = 17 }; // equal to the size of t/pid
enum { NBITS_VID = (64 - NBITS_IDX - NBITS_DIR) }; // 0: index vertex, ID: normal vertex

#define ASSOCIATIVITY 8


struct ikey_t {
uint64_t dir : NBITS_DIR; // direction
uint64_t pid : NBITS_IDX; // predicate
uint64_t vid : NBITS_VID; // vertex

    __host__ __device__
    ikey_t(): vid(0), pid(0), dir(0) { }

    __host__ __device__
    ikey_t(uint64_t v, uint64_t p, uint64_t d): vid(v), pid(p), dir(d) {
    }

    __device__
    bool operator == (const ikey_t &key) const {
        if ((vid == key.vid) && (pid == key.pid) && (dir == key.dir))
            return true;
        return false;
    }

};

// 64-bit internal pointer (size < 256M and off off < 64GB)
enum { NBITS_SIZE = 28 };
enum { NBITS_PTR = 36 };

struct iptr_t {
uint64_t size: NBITS_SIZE;
uint64_t off: NBITS_PTR;

    __device__
    iptr_t(): size(0), off(0) { }

    __device__
    iptr_t(uint64_t s, uint64_t o): size(s), off(o) {
    }
};

// 128-bit vertex (key)
struct vertex_t {
    ikey_t key; // 64-bit: vertex | predicate | direction
    iptr_t ptr; // 64-bit: size | offset
};

// 32-bit edge (value)
struct edge_t {
    uint32_t val;  // vertex ID
};

__device__
static uint64_t myhash(ikey_t lkey)
{
    uint64_t r = 0;
    r += lkey.vid;
    r <<= NBITS_IDX;
    r += lkey.pid;
    r <<= NBITS_DIR;
    r += lkey.dir;

    uint64_t key = r;
    key = (~key) + (key << 21); // key = (key << 21) - key - 1;
    key = key ^ (key >> 24);
    key = (key + (key << 3)) + (key << 8); // key * 265
    key = key ^ (key >> 14);
    key = (key + (key << 2)) + (key << 4); // key * 21
    key = key ^ (key >> 28);
    key = key + (key << 31);
    return key;
}

__device__ __forceinline__
uint64_t map_location_on_shards(uint64_t offset, uint64_t *head_list, uint64_t shard_sz)
{
    return head_list[offset / shard_sz] + offset % shard_sz;
}


/*********************************************
 *                                           *
 *                Query functions            *
 *                                           *
 *********************************************/

__global__
void d_generate_key_list_i2u(int *result_table,
                                int index_vertex,
                                int direction,
                                ikey_t *key_list,
                                int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<size) {
        ikey_t r = ikey_t(0,index_vertex,direction);
        key_list[index] = r;
    }
}


void generate_key_list_i2u(int *result_table,
                       int index_vertex,
                       int direction,
                       void *key_list,
                       int query_size,
                       hipStream_t stream_id)
{
    // int gridsize = (int) (ceil((double)query_size / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid= dim3(gridsize, 1, 1);
    d_generate_key_list_i2u<<<WUKONG_GET_BLOCKS(query_size), WUKONG_CUDA_NUM_THREADS, 0, stream_id >>>(result_table, index_vertex, direction, (ikey_t*) key_list, query_size);
}


__device__
void d_generate_key_list_k2u(int index,
                                int *result_table,
                                ikey_t *key_list,
                                int start,
                                int direction,
                                int predict,
                                int col_num,
                                int row_num)
{
    int prev_id = result_table[index * col_num - start - 1];
    ikey_t r = ikey_t(prev_id,predict,direction);
    key_list[index] = r;
}


////////////////////////////////////////////////////////////////////////////////////

__global__
void get_slot_id_list_kernel(vertex_t* d_vertex_addr,
                 ikey_t* d_key_list,
                 uint64_t* d_slot_id_list,
                 ikey_t empty_key,
                 rdf_segment_meta_t *seg_meta,
                 uint64_t* vertex_headers,
                 uint64_t pred_vertex_shard_size,
                 int query_size)

{

    // int index = blockIdx.x * blockDim.x * blockDim.y
                // + threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < query_size) {
        ikey_t key =  d_key_list[index];
        uint64_t bucket_id=map_location_on_shards(myhash(key) % seg_meta->num_buckets,
                                                  vertex_headers,
                                                  pred_vertex_shard_size);
        while (true) {
            for (uint64_t i=0;i<ASSOCIATIVITY;i++) {
                uint64_t slot_id=bucket_id*ASSOCIATIVITY+i;
                if (i < ASSOCIATIVITY - 1) {
                    // data part
                    if (d_vertex_addr[slot_id].key == d_key_list[index]) {
                        // we found it
                        d_slot_id_list[index] = slot_id;
                        return;
                    }
                } else {
                    if (!(d_vertex_addr[slot_id].key == empty_key)) {
                        // next pointer
                        // uint64_t next_bucket_id = d_vertex_addr[slot_id].key.vid-pred_metas[key.pid].indrct_hdr_start+pred_metas[key.pid].partition_sz;
                        uint64_t next_bucket_id = d_vertex_addr[slot_id].key.vid - seg_meta->ext_bucket_list[0].start + seg_meta->num_buckets;
                        bucket_id = map_location_on_shards(next_bucket_id,
                                                         vertex_headers,
                                                         pred_vertex_shard_size);
                        break;
                    } else {
                        d_slot_id_list[index] = (uint64_t)(-1);
                        return;
                    }
                }
            }
        }
    }
}


__device__
void d_get_slot_id_list(int index,
                vertex_t* d_vertex_addr,
                ikey_t* d_key_list,
                uint64_t* d_slot_id_list,
                ikey_t empty_key,
                rdf_segment_meta_t *seg_meta,
                uint64_t* vertex_headers,
                uint64_t pred_vertex_shard_size,
                int query_size)
{
    ikey_t key =  d_key_list[index];
    uint64_t bucket_id=map_location_on_shards(myhash(key) % seg_meta->num_buckets,
                                              vertex_headers,
                                              pred_vertex_shard_size);
    while (true) {
        for (uint64_t i=0;i<ASSOCIATIVITY;i++) {
            uint64_t slot_id = bucket_id * ASSOCIATIVITY + i;
            if (i < ASSOCIATIVITY - 1) {
                // data part
                if (d_vertex_addr[slot_id].key == d_key_list[index]) {
                    // we found it
                    d_slot_id_list[index] = slot_id;
                    return;
                }
            } else {
                if (!(d_vertex_addr[slot_id].key == empty_key)) {
                    // next pointer
                    uint64_t next_bucket_id = d_vertex_addr[slot_id].key.vid - seg_meta->ext_bucket_list[0].start + seg_meta->num_buckets;
                    bucket_id = map_location_on_shards(next_bucket_id,
                                                     vertex_headers,
                                                     pred_vertex_shard_size);
                    break;
                } else {
                    d_slot_id_list[index] = (uint64_t)(-1);
                    return;
                }
            }
        }
    }
}


// void get_slot_id_list(void* d_vertex_addr,
                 // void* d_key_list,
                 // uint64_t* d_slot_id_list,
                 // pred_meta_t* pred_metas,
                 // uint64_t* vertex_headers,
                 // uint64_t pred_vertex_shard_size,
                 // int query_size,
                 // hipStream_t stream_id)

// done
void gpu_get_slot_id_list(GPUEngineParam &param, hipStream_t stream)
{
    // int gridsize = (int) (ceil((double)param.query.row_num / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid = dim3(gridsize, 1, 1);

    ikey_t empty_key = ikey_t();

    get_slot_id_list_kernel<<<WUKONG_GET_BLOCKS(param.query.row_num),
        WUKONG_CUDA_NUM_THREADS, 0, stream>>>(
            param.gpu.d_vertex_addr,
            param.gpu.d_key_list,   // (ikey_t*)d_key_list,
            param.gpu.d_slot_id_list,  // d_slot_id_list,
            empty_key, // empty_key,
            param.gpu.d_segment_meta,// pred_metas,
            param.gpu.d_vertex_mapping, // vertex_headers,
            param.gpu.vertex_block_sz, // pred_vertex_shard_size,
            param.query.row_num); // query_size);
}

__global__
void get_edge_list_kernel(uint64_t *slot_id_list,
                    vertex_t *d_vertex_addr,
                    int *index_list,
                    int *index_list_mirror,
                    uint64_t *off_list,
                    uint64_t pred_orin_edge_start,
                    uint64_t* edge_headers,
                    uint64_t pred_edge_shard_size,
                    int query_size)
{
    // int index = blockIdx.x * blockDim.x * blockDim.y
                // + threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<query_size)
    {
        uint64_t id = slot_id_list[index];
        iptr_t r = d_vertex_addr[id].ptr;
        //if (index<10)
        //printf("r.size:%d\n",r.size);
        index_list_mirror[index] = r.size;
        //off_list[index] = map_location_on_shards(r.off-pred_orin_edge_start,
        //                                         edge_headers,
        //                                         pred_edge_shard_size);
        off_list[index] = r.off-pred_orin_edge_start;
   }
}


__device__
void d_get_edge_list(int index,
                    uint64_t *slot_id_list,
                    vertex_t *d_vertex_addr,
                    int *index_list,
                    int *index_list_mirror,
                    uint64_t *off_list,
                    uint64_t pred_orin_edge_start,
                    uint64_t* edge_headers,
                    uint64_t pred_edge_shard_size,
                    int query_size)
{
    uint64_t id = slot_id_list[index];
    iptr_t r = d_vertex_addr[id].ptr;
    //if (index<10)
    //printf("r.size:%d\n",r.size);
    index_list_mirror[index] = r.size;
    //off_list[index] = map_location_on_shards(r.off-pred_orin_edge_start,
    //                                         edge_headers,
    //                                         pred_edge_shard_size);
    off_list[index] = r.off-pred_orin_edge_start;

}


// void get_edge_list(uint64_t *slot_id_list,
                    // void *d_vertex_addr,
                    // int *index_list,
                    // int *index_list_mirror,
                    // uint64_t *ptr_list,
                    // uint64_t pred_orin_edge_start,
                    // uint64_t* edge_headers,
                    // uint64_t pred_edge_shard_size,
                    // int query_size,
                    // hipStream_t stream_id)
// done
void gpu_get_edge_list(GPUEngineParam &param, hipStream_t stream_id)
{
    // int gridsize = (int) (ceil((double) param.query.row_num / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid= dim3(gridsize, 1, 1);

    get_edge_list_kernel<<<WUKONG_GET_BLOCKS(param.query.row_num),
        WUKONG_CUDA_NUM_THREADS, 0, stream_id>>>(
                    param.gpu.d_slot_id_list,
                    param.gpu.d_vertex_addr, // (vertex_t*)d_vertex_addr,
                    param.gpu.d_prefix_sum_list, // index_list,
                    param.gpu.d_edge_size_list, // index_list_mirror,
                    param.gpu.d_offset_list, // ptr_list,
                    param.query.segment_edge_start, // pred_orin_edge_start,
                    param.gpu.d_edge_mapping, // edge_headers,
                    param.gpu.edge_block_sz, // pred_edge_shard_size,
                    param.query.row_num); // query_size);

}



__global__
void k_get_edge_list_k2k(uint64_t *slot_id_list,
                    vertex_t *d_vertex_addr,
                    int *index_list,
                    int *index_list_mirror,
                    uint64_t *ptr_list,
                    int query_size,
                    edge_t *edge_addr,
                    int *result_table,
                    int col_num,
                    int end,
                    uint64_t pred_orin_edge_start,
                    uint64_t* edge_headers,
                    uint64_t pred_edge_shard_size)
{
    // int index = blockIdx.x * blockDim.x * blockDim.y
                // + threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<query_size)
    {
        uint64_t id = slot_id_list[index];
        iptr_t r = d_vertex_addr[id].ptr;

        index_list_mirror[index] = 0;

        int end_id = result_table[index * col_num - end - 1];
        ptr_list[index] = r.off-pred_orin_edge_start;
        for(int k=0;k<r.size;k++){
            uint64_t ptr = map_location_on_shards(r.off-pred_orin_edge_start+k,
                                                  edge_headers,
                                                  pred_edge_shard_size);

            if (edge_addr[ptr].val==end_id)
            {
                index_list_mirror[index] = 1;
                break;
            }
        }
   }


}

// void get_edge_list_k2k(uint64_t *slot_id_list,
                    // void *d_vertex_addr,
                    // int *index_list,
                    // int *index_list_mirror,
                    // uint64_t *ptr_list,
                    // int query_size,
                    // void *edge_addr,
                    // int *result_table,
                    // int col_num,
                    // int end,
                    // uint64_t pred_orin_edge_start,
                    // uint64_t* edge_headers,
                    // uint64_t pred_edge_shard_size,
                    // hipStream_t stream_id)

void gpu_get_edge_list_k2k(GPUEngineParam &param, hipStream_t stream)
{
    // int gridsize = (int) (ceil((double)param.query.row_num / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid = dim3(gridsize, 1, 1);

    k_get_edge_list_k2k<<<WUKONG_GET_BLOCKS(param.query.row_num),
        WUKONG_CUDA_NUM_THREADS, 0, stream>>>(
                    param.gpu.d_slot_id_list, // slot_id_list,
                    param.gpu.d_vertex_addr, // (vertex_t*)d_vertex_addr,
                    param.gpu.d_prefix_sum_list, // index_list,
                    param.gpu.d_edge_size_list, // index_list_mirror,
                    param.gpu.d_offset_list, // ptr_list,
                    param.query.row_num, // query_size,
                    param.gpu.d_edge_addr, // (edge_t*)edge_addr,
                    param.gpu.d_in_rbuf, // result_table,
                    param.query.col_num, // col_num,
                    param.query.end_vid, // end,
                    param.query.segment_edge_start, // pred_orin_edge_start,
                    param.gpu.d_edge_mapping, // edge_headers,
                    param.gpu.edge_block_sz); // pred_edge_shard_size);

}


__global__
void k_get_edge_list_k2c(uint64_t *slot_id_list,
                    vertex_t *d_vertex_addr,
                    int *index_list,
                    int *index_list_mirror,
                    uint64_t *ptr_list,
                    int query_size,
                    edge_t *edge_addr,
                    int end,
                    uint64_t pred_orin_edge_start,
                    uint64_t* edge_headers,
                    uint64_t pred_edge_shard_size)
{
    // int index = blockIdx.x * blockDim.x * blockDim.y
                // + threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<query_size)
    {
        uint64_t id = slot_id_list[index];
        iptr_t r = d_vertex_addr[id].ptr;

        index_list_mirror[index] = 0;
        ptr_list[index] =r.off-pred_orin_edge_start;
        for(int k=0;k<r.size;k++){
            uint64_t ptr = map_location_on_shards(r.off-pred_orin_edge_start+k,
                                                  edge_headers,
                                                  pred_edge_shard_size);
            if (edge_addr[ptr].val==end)
            {
                index_list_mirror[index] = 1;
                break;
            }
        }
   }


}

// void get_edge_list_k2c(uint64_t *slot_id_list,
                    // void *d_vertex_addr,
                    // int *index_list,
                    // int *index_list_mirror,
                    // uint64_t *ptr_list,
                    // int query_size,
                    // void *edge_addr,
                    // int end,
                    // uint64_t pred_orin_edge_start,
                    // uint64_t* edge_headers,
                    // uint64_t pred_edge_shard_size,
                    // hipStream_t stream_id)


void gpu_get_edge_list_k2c(GPUEngineParam &param, hipStream_t stream)
{
    // int gridsize = (int) (ceil((double)param.query.row_num / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid= dim3(gridsize, 1, 1);

    k_get_edge_list_k2c<<<WUKONG_GET_BLOCKS(param.query.row_num),
        WUKONG_CUDA_NUM_THREADS, 0, stream>>>(
                    param.gpu.d_slot_id_list,
                    param.gpu.d_vertex_addr,
                    param.gpu.d_prefix_sum_list, // index_list,
                    param.gpu.d_edge_size_list, //index_list_mirror,
                    param.gpu.d_offset_list, // ptr_list,
                    param.query.row_num,    // query_size,
                    param.gpu.d_edge_addr, // (edge_t*)edge_addr,
                    param.query.end_vid, // end,
                    param.query.segment_edge_start, // pred_orin_edge_start,
                    param.gpu.d_edge_mapping, // edge_headers,
                    param.gpu.edge_block_sz); // pred_edge_shard_size);

}

// done
__global__
void k_update_result_buf_i2u(int *result_table,
                                  int *updated_result_table,
                                  int *index_list,
                                  uint64_t *ptr_list,
                                  edge_t *edge_addr,
                                  uint64_t* edge_headers,
                                  uint64_t pred_edge_shard_size)
{
    // int index = blockIdx.x * blockDim.x * blockDim.y
                // + threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int edge_num = 0;
    edge_num = index_list[0];

    if(index<edge_num) {
            uint64_t ptr = map_location_on_shards(ptr_list[0]+index,
                                                  edge_headers,
                                                  pred_edge_shard_size);
            //printf("ptr:%d\n",(&(edge_addr[ptr])+index)->val);
            updated_result_table[index] = edge_addr[ptr].val;
    }

}

// int update_result_table_i2u(int *result_table,
                                  // int *updated_result_table,
                                  // int *index_list,
                                  // uint64_t *ptr_list,
                                  // void *edge_addr,
                                  // uint64_t* edge_headers,
                                  // uint64_t pred_edge_shard_size,
                                  // hipStream_t stream_id)

// done
int gpu_update_result_buf_i2u(GPUEngineParam& param, hipStream_t stream)
{
    int table_size = 0;//index_list[query_size-1];
    CUDA_ASSERT(hipMemcpyAsync(&table_size,
               param.gpu.d_prefix_sum_list,
               sizeof(int),
               hipMemcpyDeviceToHost, stream));


    // int gridsize = (int) (ceil((double)table_size / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid= dim3(gridsize, 1, 1);

    k_update_result_buf_i2u<<<WUKONG_GET_BLOCKS(param.query.row_num),
        WUKONG_CUDA_NUM_THREADS, 0, stream>>>(param.gpu.d_in_rbuf,
         param.gpu.d_out_rbuf,
         param.gpu.d_prefix_sum_list, //index_list,
         param.gpu.d_offset_list, // ptr_list,
         param.gpu.d_edge_addr,  // (edge_t*)edge_addr,
         param.gpu.d_edge_mapping, // edge_headers,
         param.gpu.edge_block_sz); // pred_edge_shard_size);

    CUDA_ASSERT( hipStreamSynchronize(stream) );
    return table_size;
}





// done
__global__
void k_update_result_buf_k2k(int *result_table,
                                  int *updated_result_table,
                                  int *index_list,
                                  uint64_t *ptr_list,
                                  int column_num,
                                  edge_t *edge_addr,
                                  int end,
                                  int query_size)
{
    // int index = blockIdx.x * blockDim.x * blockDim.y
                // + threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index<query_size) {
        int edge_num = 0,start=0;
        if(index==0) {
            edge_num = index_list[index];
            start = 0;
        }
        else {
            edge_num = index_list[index] - index_list[index - 1];
            start = column_num*index_list[index - 1];
        }
        int buff[20];
        for(int c=0;c<column_num;c++){
            buff[c] = result_table[column_num*index+c];
        }
        for(int k=0;k<edge_num;k++){
            for(int c=0;c<column_num;c++){
                updated_result_table[start+c] = buff[c];//result_table[column_num*index+c];
            }
        }
    }
}
// TODO
// int update_result_table_k2k(int *result_table,
                                  // int *updated_result_table,
                                  // int *index_list,
                                  // uint64_t *ptr_list,
                                  // int column_num,
                                  // void *edge_addr,
                                  // int end,
                                  // int query_size,
                                  // hipStream_t stream_id)
// done
int gpu_update_result_buf_k2k(GPUEngineParam& param, hipStream_t stream)
{
    // Siyuan: index_list[query_size-1]是新的table的行数
    int table_size = 0;//index_list[query_size-1];
    CUDA_ASSERT(hipMemcpyAsync(&table_size,
               param.gpu.d_prefix_sum_list + param.query.row_num - 1,
               sizeof(int),
               hipMemcpyDeviceToHost, stream));

    // int gridsize = (int) (ceil((double)query_size / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid= dim3(gridsize, 1, 1);

    k_update_result_buf_k2k<<<WUKONG_GET_BLOCKS(param.query.row_num),
        WUKONG_CUDA_NUM_THREADS, 0, stream>>>(
         param.gpu.d_in_rbuf,//   result_table,
         param.gpu.d_out_rbuf,// updated_result_table,
         param.gpu.d_prefix_sum_list,// index_list,
         param.gpu.d_offset_list, // ptr_list,
         param.query.col_num, // column_num,
         param.gpu.d_edge_addr,//  (edge_t*)edge_addr,
         param.query.end_vid, // end,
         param.query.row_num);

    CUDA_ASSERT( hipStreamSynchronize(stream) );
    return table_size * param.query.col_num;
}


// done
void gpu_calc_prefix_sum(GPUEngineParam& param,
                     hipStream_t stream)
{
    thrust::device_ptr<int> d_in_ptr(param.gpu.d_prefix_sum_list);
    thrust::device_ptr<int> d_out_ptr(param.gpu.d_edge_size_list);
    thrust::inclusive_scan(thrust::cuda::par.on(stream), d_in_ptr, d_in_ptr + param.query.row_num, d_out_ptr);
}


// Siyuan: 计算history中每条record(每行)的目的地server id
// done
__global__
void hash_tuples_to_server(int *result_table,
                                  int *server_id_list,
                                  int start,
                                  int col_num,
                                  int num_sub_request,
                                  int query_size)
{
    // int index = blockIdx.x * blockDim.x * blockDim.y
                // + threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<query_size) {
        // Siyuan: index指定是第几行
        server_id_list[index] =  result_table[index * col_num + (-start - 1)] % num_sub_request;
    }
}

// done
void hash_dispatched_server_id(int *result_table,
                                  int *server_id_list,
                                  int start,
                                  int col_num,
                                  int num_sub_request,
                                  int query_size,
                                  hipStream_t stream)
{
    // int gridsize = (int) (ceil((double)query_size / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid= dim3(gridsize, 1, 1);

    hash_tuples_to_server<<<WUKONG_GET_BLOCKS(query_size),
        WUKONG_CUDA_NUM_THREADS, 0, stream>>>(result_table,
                                  server_id_list,
                                  start,
                                  col_num,
                                  num_sub_request,
                                  query_size);
}

__global__
void k_history_dispatch(int *result_table,
                        int* position_list,
                        int* server_id_list,
                        int* server_sum_list,
                        int start,
                        int col_num,
                        int num_sub_request,
                        int query_size)
{
    // int index = blockIdx.x * blockDim.x * blockDim.y
                // + threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<query_size) {
        int server_id =server_id_list[index];
        position_list[index] = atomicAdd(&server_sum_list[server_id],1);
    }

}

// done
void history_dispatch(int *result_table,
                        int* position_list,
                        int* server_id_list,
                        int* server_sum_list,
                        int start,
                        int col_num,
                        int num_sub_request,
                        int query_size,
                        hipStream_t stream)
{
    // int gridsize = (int) (ceil((double)query_size / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid= dim3(gridsize, 1, 1);

    k_history_dispatch<<<WUKONG_GET_BLOCKS(query_size),
        WUKONG_CUDA_NUM_THREADS, 0, stream>>>(result_table,
                                               position_list,
                                               server_id_list,
                                               server_sum_list,
                                               start,
                                               col_num,
                                               num_sub_request,
                                               query_size);
}


// Siyuan: updated_result_table是一个device上的大buffer，
// parent history table通过不同的偏移量把sub query table映射到这个buffer中
// done
__global__
void k_split_result_buf(int *d_in_result_buf,
                                  int *d_out_result_buf,
                                  int *d_position_list,
                                  int *server_id_list,
                                  int *sub_table_hdr_list,
                                  int column_num,
                                  int num_sub_request,
                                  int query_size)
{
    // int index = blockIdx.x * blockDim.x * blockDim.y
                // + threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < query_size) {
        int dst_sid = server_id_list[index];
        int mapped_index = sub_table_hdr_list[dst_sid] + d_position_list[index];
        for (int c = 0; c < column_num; c++) {
            d_out_result_buf[column_num * mapped_index + c] = d_in_result_buf[column_num * index + c];
        }
    }
}

// done
void gpu_split_result_buf(GPUEngineParam &param, int num_servers, hipStream_t stream)
{
    // bowrrow other buffers for temporary use
    int *d_position_list = (int*) param.gpu.d_slot_id_list;
    int *d_server_id_list = param.gpu.d_prefix_sum_list;
    int *d_server_sum_list = param.gpu.d_edge_size_list;

    // int gridsize = (int) (ceil((double)query_size / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid= dim3(gridsize, 1, 1);

    k_split_result_buf<<<WUKONG_GET_BLOCKS(param.query.row_num),
        WUKONG_CUDA_NUM_THREADS, 0, stream>>>(
            param.gpu.d_in_rbuf,
            param.gpu.d_out_rbuf,
            d_position_list,
            d_server_id_list,
            d_server_sum_list,
            param.query.col_num, //column_num,
            num_servers, // num_sub_request,
            param.query.row_num);//query_size);
}


// done
void gpu_shuffle_result_buf(GPUEngineParam& param, vector<int>& buf_sizes, hipStream_t stream)
{
    // bowrrow other buffers for temporary use
    int *d_position_list = (int*) param.gpu.d_slot_id_list;
    int *d_server_id_list = param.gpu.d_prefix_sum_list;
    int *d_server_sum_list = param.gpu.d_edge_size_list;

    int num_jobs = buf_sizes.size();

    // Siyuan: 计算每条record将要被发送到的server id
    hash_dispatched_server_id(param.gpu.d_in_rbuf,
                                  d_server_id_list,
                                  param.query.start_vid,
                                  param.query.col_num,
                                  num_jobs,
                                  param.query.row_num,
                                  stream);

    // TODO
    // Siyuan: 此处是把parent history table切分成child history table
    history_dispatch(param.gpu.d_in_rbuf,
                     d_position_list,
                         d_server_id_list,
                         d_server_sum_list,
                         param.query.start_vid,
                         param.query.col_num,
                         num_jobs,
                         param.query.row_num,
                         stream);

    // Siyuan: gpu_sub_table_size_list中存的是每个sub table的
    CUDA_ASSERT(hipMemcpyAsync(&buf_sizes[0],
                                  d_server_sum_list,
                                  sizeof(int) * num_jobs,
                                  hipMemcpyDeviceToHost,
                                  stream));

    CUDA_ASSERT(hipStreamSynchronize(stream));

    // Siyuan: 对d_server_sum_list计算exclusive的前置和
    thrust::device_ptr<int> d_server_sum_list_ptr(d_server_sum_list);
    thrust::exclusive_scan(thrust::cuda::par.on(stream), d_server_sum_list_ptr, d_server_sum_list_ptr + num_jobs, d_server_sum_list_ptr);
    // 函数返回之后d_server_sum_list中就是[0,5,12]这样的前值和
}


__global__
void lookup_hashtable_k2u(GPUEngineParam param)
{

    // int index = blockIdx.x * blockDim.x * blockDim.y
                // + threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= param.query.row_num)
        return;

    d_generate_key_list_k2u(index,
            param.gpu.d_in_rbuf, //result_table,
            param.gpu.d_key_list,
            param.query.start_vid, // start,
            param.query.dir,// direction,
            param.query.pid,// predict,
            param.query.col_num,
            param.query.row_num);

    // get_slot_id_list
    ikey_t empty_key = ikey_t();

    d_get_slot_id_list(index,
            param.gpu.d_vertex_addr,
            param.gpu.d_key_list,
            param.gpu.d_slot_id_list,
            empty_key,
            param.gpu.d_segment_meta, // pred_metas,
            param.gpu.d_vertex_mapping,
            param.gpu.vertex_block_sz,
            param.query.row_num);


    // get_edge_list
    d_get_edge_list(index,
            param.gpu.d_slot_id_list,
            param.gpu.d_vertex_addr,
            param.gpu.d_prefix_sum_list,
            param.gpu.d_edge_size_list,
            param.gpu.d_edge_off_list,
            param.query.segment_edge_start, // pred_orin_edge_start,
            param.gpu.d_edge_mapping,// edge_headers,
            param.gpu.edge_block_sz, // pred_edge_shard_size,
            param.query.row_num); // query_size);
}



void gpu_lookup_hashtable_k2u(GPUEngineParam& param, hipStream_t stream = 0)
{

    // CHECK_HANDLER_ARG(arg);

    // int gridsize = (int) (ceil((double)arg.query_size / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid= dim3(gridsize, 1, 1);

    lookup_hashtable_k2u<<<WUKONG_GET_BLOCKS(param.query.row_num),
        WUKONG_CUDA_NUM_THREADS, 0, stream>>>(param);
}



// done
__global__
void d_update_result_table_k2u(int *result_table,
                                  int *updated_result_table,
                                  int *index_list,
                                  uint64_t *off_list,
                                  edge_t *edge_addr,
                                  uint64_t* edge_headers,
                                  uint64_t pred_edge_shard_size,
                                  int column_num,
                                  int query_size)
{
    // int index = blockIdx.x * blockDim.x * blockDim.y
                // + threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //int index = full_index/200/(column_num+1);
    if(index<query_size) {
        //extern __shared__ int result_matrix[];

        int edge_num = 0,start=0;
        if(index==0) {
            edge_num = index_list[index];
            start = 0;
        }
        else {
            edge_num = index_list[index] - index_list[index - 1];
            start = (column_num+1)*index_list[index - 1];
        }

        int buff[20];
        for(int c=0;c<column_num;c++){
            buff[c] = result_table[column_num*index+c];
        }

        for(int k=0;k<edge_num;k++){
            // #1 把原有的列放到new history table中
            for(int c=0;c<column_num;c++){
                updated_result_table[start+k*(column_num+1)+c] = buff[c];//result_table[column_num*index+c];
            }
            // #2 把新的列放到new history table里
            uint64_t ptr = map_location_on_shards(off_list[index]+k,
                                                  edge_headers,
                                                  pred_edge_shard_size);

            updated_result_table[start+k*(column_num+1)+column_num] = edge_addr[ptr].val;
        }
    }

}

// update_result_table_k2u
int gpu_update_result_buf_k2u(GPUEngineParam& param, hipStream_t stream)
{

    int table_size = 0;//index_list[query_size-1];
    CUDA_ASSERT( hipMemcpyAsync(&table_size,
               param.gpu.d_prefix_sum_list + param.query.row_num - 1,
               sizeof(int),
               hipMemcpyDeviceToHost, stream) );

    //query_size = query_size*200*(column_num+1);
    // int gridsize = (int) (ceil((double)query_size / (blocksize * blocksize)));
    // dim3 dimBlock = dim3(blocksize, blocksize, 1);
    // dim3 dimGrid= dim3(gridsize, 1, 1);

    // auto d_vertex_mapping = thrust::raw_pointer_cast( &(param.gpu.vertex_mapping_dv[0]) );
    // auto d_edge_mapping = thrust::raw_pointer_cast( &(param.gpu.edge_mapping_dv[0]) );

    d_update_result_table_k2u<<<WUKONG_GET_BLOCKS(param.query.row_num),
        WUKONG_CUDA_NUM_THREADS, 0, stream>>>(
                param.gpu.d_in_rbuf,
                param.gpu.d_out_rbuf,
                param.gpu.d_prefix_sum_list,
                param.gpu.d_offset_list,
                param.gpu.d_edge_addr, // (edge_t*)edge_addr,
                param.gpu.d_edge_mapping, //edge_headers,
                param.gpu.edge_block_sz, // pred_edge_shard_size,
                param.query.col_num,//column_num,
                param.query.row_num//query_size
         );

    CUDA_ASSERT( hipStreamSynchronize(stream) );
    return table_size*(param.query.col_num + 1);

}




